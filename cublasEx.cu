#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// metrics
#include <sys/time.h>

int main(int argc, char *argv[])
{
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    double *a, *v;
    double *dev_a;
    double *dev_v;
    double *dev_z;

    int N = 1024;
    int matched;
    if (argc > 1) {
        matched = sscanf(argv[1], "%d", &N);
        if (matched < 1) {
            printf("Usage: %s number\n", argv[0]);
            return -1;
        }
    }

    printf("Matrix size: %dx%d\n", N, N);
    a = (double *) malloc (N * N * sizeof(double));
    v = (double *) malloc (N * sizeof(double));
    hipMalloc((void **)&dev_a, N * N * sizeof(double));
    hipMalloc((void **)&dev_v, N * sizeof(double));
    hipMalloc((void **)&dev_z, N * sizeof(double));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i*N + j] = (i-1) * N + 2*j;
        }
    }

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("handle creation failed");
    }

    double alpha, beta;
    alpha = 1.0;
    beta = 0.0;
    stat = hipblasSetMatrix(N, N, sizeof(double), a, N, dev_a, N);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("set matrix fail");
    }

    for (int i = 0; i < N; i++) {
        v[i] = 1.1f;
    }

    stat = hipblasSetVector(N, sizeof(double), v, 1, dev_v, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("set vector fail");
    }

    for (int i = 0; i < N; i++) {
        v[i] = 0;
    }

    stat = hipblasSetVector(N, sizeof(double), v, 1, dev_z, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("set vector fail");
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    stat = hipblasDgemv(handle, HIPBLAS_OP_T, N, N, &alpha, dev_a, N, dev_v, 1, &beta, dev_z, 1);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("dgemvv fail");
    }

    /*for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", a[i*N+j]);
        }
        printf("\n");
    }
    printf("\n");*/
    
    stat = hipblasGetVector(N, sizeof(double), dev_z, 1, v, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("download fail");
    }

    /*for (int i = 0; i < N; i++) {*/
        /*printf("%f ", v[i]);*/
    /*}*/

    float elapsedMs = 0.0f;
    hipEventElapsedTime(&elapsedMs, start, stop);
    printf("Time elapsed: %fms\n", elapsedMs);

    hipFree(dev_a);
    hipFree(dev_v);
    hipFree(dev_z);
    hipblasDestroy(handle);
    free(a);
    free(v);
    return 0;
}
