#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "common.h"
#include "plainKernel.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void matVectorMul(double* mat, double* vec, double *res, sizeInfo size)
{
    int rowId = threadIdx.x + blockIdx.x * blockDim.x;
    double sum = 0.0;

    if (rowId < size.rows) {
        for (int i = 0; i < size.cols; i++) {
            sum += mat[rowId * size.cols + i] * vec[i];
        }
        res[rowId] = sum;
    }
}

void matVectorMulHost(double *mat, double *vec, double *res, sizeInfo size)
{
    double sum = 0.0;

    for (int i = 0; i < size.rows; i++) {
        sum = 0.0;
        for (int j = 0; j < size.cols; j++) {
            sum += mat[i * size.cols + j] * vec[j];
        }
        res[i] = sum;
    }
}

float plainKernelSetup(int rows, int cols, bool runCPU)
{
    double *matrix, *v, *result, *result_cpu;
    double *dev_matrix, *dev_v, *dev_result;
    sizeInfo sizes;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    sizes.rows = rows;
    sizes.cols = cols;

    std::cout << "\nRunning MV multiplication for a " << rows << "x" << cols << " matrix..." << std::endl;

    int matrixSize = rows * cols * sizeof(double);
    int vectorSize = cols * sizeof(double);
    int resultSize = rows * sizeof(double);

    // allocate matrix & vector memory
    matrix = (double *) malloc(matrixSize);
    v = (double *) malloc(vectorSize);
    result = (double *) malloc(resultSize);
    result_cpu = (double *) malloc(resultSize);

    // allocate cuda memory
    gpuErrchk( hipMalloc(&dev_matrix, matrixSize) );
    gpuErrchk( hipMalloc(&dev_v, vectorSize) );
    gpuErrchk( hipMalloc(&dev_result, resultSize) );

    // randomize matrix elements
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            /*matrix[i * cols + j] = rand() / (RAND_MAX * 1.0) * 2.0 - 1.0;*/
            matrix[i * cols + j] = ceil(rand() / (RAND_MAX * 1.0) * 10.0);
        }
    }

    // randomize vector elements
    for (int i = 0; i < cols; i++) {
        v[i] = rand() / (RAND_MAX * 1.0) * 2.0 - 1.0;
        /*v[i] = ceil(rand() / (RAND_MAX * 1.0) * 10.0);*/
    }

    // copy from host to device
    gpuErrchk( hipMemcpy(dev_matrix, matrix, matrixSize, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dev_v, v, vectorSize, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemset(dev_result, 0, resultSize) );

    gpuErrchk( hipEventRecord(start) );
    matVectorMul<<<rows / 16 + 1, 16>>>(dev_matrix, dev_v, dev_result, sizes);
    gpuErrchk( hipEventRecord(stop) );

    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk( hipMemcpy(result, dev_result, resultSize, hipMemcpyDeviceToHost) );

    gpuErrchk( hipEventSynchronize(stop) );
    float msec = 0.0f;
    gpuErrchk( hipEventElapsedTime(&msec, start, stop) );
    std::cout << msec << "ms elapsed for kernel." << std::endl;

    if (runCPU) {
        // run same multiplication on CPU
        matVectorMulHost(matrix, v, result_cpu, sizes);
    }

    gpuErrchk( hipFree(dev_matrix) );
    gpuErrchk( hipFree(dev_v) );
    gpuErrchk( hipFree(dev_result) );

    free(matrix);
    free(v);
    free(result);
    free(result_cpu);

    return msec;
}
